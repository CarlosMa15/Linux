#include "hip/hip_runtime.h"
#include <stdio.h>


#define __min(x,y) ((x)<(y)?(x):(y))

;
#define N 1024

extern int hipMemcpy();					//Memcopy
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();				//Memcopy
extern __global__ void mv_GPU(float *, float (*)[N], float *);  //Kernal

int compare(float *a, float *b, int size, double threshold) {
    printf("CHECK2\n");
    int i;
    for (i=0; i<size; i++) {
      if (abs(a[i]-b[i]) > threshold) return 0;
    }
    return 1;
}

void normalMV(int *ptr, int nr, float *t, float *data, int *indices, float *b){
  int i;
  int j;
  for (i=0; i<nr; i++) {
     for (j = ptr[i]; j<ptr[i+1]; j++) {
        t[i] = t[i] + data[j] * b[indices[j]];
     }
  }

  /************************************************
  int i;
  int j;
  for(i=0;i<N;i++)
    for(j=0;j<N;j++)
		a[i] = a[i] +  c[j*N+i] * b[j] ;
  ************************************************/
}

extern __global__ void mv_GPU(int *ptr, int nr, float *t, float *data, int *indices, float *b)
  {

    int i;
    int j;
    for (i=0; i<nr; i++) {
       for (j = ptr[i]; j<ptr[i+1]; j++) {
          t[i] = t[i] + data[j] * b[indices[j]];
       }
    }

    /*******************************************************************
    int bx;
    int tx;
    float acpy;
    __shared__ float bcpy[32];
    int k;
    int j;

    bx = blockIdx.x;
    tx = threadIdx.x;
    int i = bx * blockDim.x + tx;
    int tiles = (N+blockDim.x-1)/blockDim.x;

    if (tx <= -(blockDim.x * bx) + N-1)
      {
        acpy = 0.0;
	// suif_tmp0 = ((float *)(float (*)[])a)[i];
      
    for (k = 0; k <= tiles; k++)
      {
          bcpy[tx] = b[blockDim.x * k + tx];
          __syncthreads();
          for (j = 32 * k; j <= __min(32 * k + 31, N-1); j++)
          {
              acpy = acpy + c[j][i] * bcpy[j - 32 * k];
          }
        __syncthreads();
      }
        a[i] = acpy;
      }
      ****************************************************************/
  }

main (int argc, char **argv) {
  FILE *fp;
  char line[1024];
  int *ptr, *indices;
  float *data, *bb, *t;
  int i;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix  
  
  // Open input file and read to end of comments
  if (argc !=2) abort();
  if ((fp = fopen(argv[1], "r")) == NULL) {
     abort();
  }
  
  fgets(line, 128, fp);
  while (line[0] == '%') {
     fgets(line, 128, fp);
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, bb and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  bb = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;
  for (i=0; i<n; i++) {
     int r;
     fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));
     indices[i]--;  // start numbering at 0
     if (r!=lastr) {
     	ptr[r-1] = i;
     	lastr = r;
     }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data 
  for (i=0; i<nr; i++) {
     t[i] = 0.0;
  }
  for (i=0; i<nc; i++) {
     bb[i] = (float) rand()/1111111111;
  }

  // create CUDA event handles for timing purposes
  hipEvent_t start_event, stop_event;
  float elapsed_time_seq, elapsed_time_gpu;

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);   
  normalMV(ptr,nr,t,data,indices, bb);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_seq,start_event, stop_event);

    int *devO1Ptr;					// int * ptr
    int *devI1Ptr;				      	// int * indices
    float *devI2Ptr;				      	// float * data
    float *devI3Ptr;				      	// float * b
    float *devI4Ptr;				      	// float * t

    hipMalloc((void **)&devO1Ptr, (nr+1)*sizeof(int));
    hipMemcpy(devO1Ptr, ptr, (nr+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI1Ptr, n*sizeof(int));
    hipMemcpy(devI1Ptr,indices,n*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI2Ptr,n*sizeof(float));
    hipMemcpy(devI2Ptr,data,n*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI3Ptr,nc*sizeof(float));
    hipMemcpy(devI3Ptr,bb,nc*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&devI4Ptr,nr*sizeof(float));
    hipMemcpy(devI4Ptr, t,nr*sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid((N+31)/32, 1);
    dim3 dimBlock(32, 1);

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  // int *ptr, int nr, float *t, float *data, int *indices, float *b
  mv_GPU<<<dimGrid,dimBlock>>>(devO1Ptr,nr,devI4Ptr,devI2Ptr,devI1Ptr,devI3Ptr);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  float *d_a = NULL;
  hipMemcpy(d_a,devI4Ptr,nr*sizeof(float), hipMemcpyDeviceToHost);	   //Memcopy
  hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event);


    hipFree(devO1Ptr);
    hipFree(devI1Ptr);
    hipFree(devI2Ptr);
    hipFree(devI3Ptr);
    hipFree(devI4Ptr);

  printf("CHECK1\n");
  int res = compare( t, d_a, nr, 0.001);
  if (res == 1) {
    printf("VALID!\n  Sequential Time: %.2f msec\n  Parallel Time: %.2f msec\n Speedup = %.2f\n", 
       elapsed_time_seq, elapsed_time_gpu, elapsed_time_seq/elapsed_time_gpu);
  }
  else printf("INVALID...\n");
}
